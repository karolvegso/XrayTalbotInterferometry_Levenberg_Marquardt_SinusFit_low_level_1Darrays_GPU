#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include <fstream>
//#include <cmath>
#include <chrono>

#define _USE_MATH_DEFINES

#define M_PI 3.14159265358979323846

using namespace std;

__device__ int levmar_sinus(double* t_data_inp, double* y_data_inp, const int M_inp, double* x0_inp, double* x_fit_outp, int k_max_inp, double eps_1_inp, double eps_2_inp, double tau_inp) {
    // initial iteration variable
    int k = 0;
    int ni = 2;
    // initialize Jacobian 1D matrix
    double* J = new double[3 * M_inp];
    // fill Jacobian matrix
    for (int index_0 = 0; index_0 < M_inp; index_0++) {
        J[0 + index_0 * 3] = (-1.0f) * std::sin(t_data_inp[index_0] + x0_inp[1]);
        J[1 + index_0 * 3] = (-1.0f) * x0_inp[0] * std::cos(t_data_inp[index_0] + x0_inp[1]);
        J[2 + index_0 * 3] = -1.0f;
    }
    // initialize transpose of Jacobian matrix in 1D form
    double* J_transpose = new double[3 * M_inp];
    // fill transpose of Jacobian matrix
    for (int index_0 = 0; index_0 < M_inp; index_0++) {
        for (int index_1 = 0; index_1 < 3; index_1++) {
            J_transpose[index_0 + index_1 * M_inp] = J[index_1 + index_0 * 3];
        }
    }
    // calculate A matrix
    // initialize A matrix
    double* A = new double[9];
    // initialize A matrix to zero values
    for (int index_0 = 0; index_0 < 9; index_0++) {
        A[index_0] = 0.0f;
    }
    // calculate A matrix as J_transpose * J
    // use multiplication of 2D matrices
    for (int index_0 = 0; index_0 < 3; index_0++) {
        for (int index_1 = 0; index_1 < 3; index_1++) {
            for (int index_2 = 0; index_2 < M_inp; index_2++) {
                A[index_1 + index_0 * 3] = A[index_1 + index_0 * 3] + J_transpose[index_2 + index_0 * M_inp] * J[index_1 + index_2 * 3];
            }
        }
    }
    // calculate f function
    // initialize f function
    double* f = new double[M_inp];
    // fill f function
    for (int index_0 = 0; index_0 < M_inp; index_0++) {
        f[index_0] = y_data_inp[index_0] - x0_inp[0] * std::sin(t_data_inp[index_0] + x0_inp[1]) - x0_inp[2];
    }
    // calculate transpose of f
    // initialize transpose of f
    double* f_transpose = new double[M_inp];
    // fill transpose of f
    for (int index_0 = 0; index_0 < M_inp; index_0++) {
        f_transpose[index_0] = f[index_0];
    }
    // calculate g as J_transpose * f_transpose
    // initialize g
    double* g = new double[3];
    // initialize g to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        g[index_0] = 0.0f;
    }
    for (int index_0 = 0; index_0 < 3; index_0++) {
        for (int index_1 = 0; index_1 < 1; index_1++) {
            for (int index_2 = 0; index_2 < M_inp; index_2++) {
                g[index_1 + index_0 * 1] = g[index_1 + index_0 * 1] + J_transpose[index_2 + index_0 * M_inp] * f_transpose[index_1 + index_2 * 1];
            }
        }
    }
    // calculate norm of g
    // initialize norm of g
    double g_norm = 0.0f;
    for (int index_0 = 0; index_0 < 3; index_0++) {
        g_norm = g_norm + g[index_0] * g[index_0];
    }
    g_norm = std::sqrt(g_norm);
    // boolean variable
    bool found_bool = (g_norm <= eps_1_inp);
    // initialize mi
    double mi = 0.0f;
    double A_diag_max = 0.0f;
    if (A[0] >= A[4]) {
        A_diag_max = A[0];
    }
    else {
        A_diag_max = A[4];
    }
    if (A_diag_max >= A[8]) {
        // do nothing
    }
    else {
        A_diag_max = A[8];
    }
    // calculate mi
    mi = tau_inp * A_diag_max;

    // initialize x vector
    double* x = new double[3];
    // fill x vector
    for (int index_0 = 0; index_0 < 3; index_0++) {
        x[index_0] = x0_inp[index_0];
    }
    // initialize x_new vector
    double* x_new = new double[3];
    // initialize x_new vector to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        x_new[index_0] = 0.0f;
    }
    // initialize transpose of g
    double* g_transpose = new double[3];
    // initialize transpose of g to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        g_transpose[index_0] = 0.0f;
    }
    // initialize B matrix
    double* B = new double[9];
    // initialize B matrix to zero values
    for (int index_0 = 0; index_0 < 9; index_0++) {
        B[index_0] = 0.0f;
    }
    // initialize inversion matrix of B
    double* B_inv = new double[9];
    // initialize inversion matrix of B to zero values
    for (int index_0 = 0; index_0 < 9; index_0++) {
        B_inv[index_0] = 0.0f;
    }
    // initialize adjoint matrix of B
    double* B_adj = new double[9];
    // initialize adjoint matrix of B to zero values
    for (int index_0 = 0; index_0 < 9; index_0++) {
        B_adj[index_0] = 0.0f;
    }
    // initialize value of determinant of B
    double B_det = 0.0f;
    // initialize minors of matrix B
    double B_minor_11 = 0.0f;
    double B_minor_12 = 0.0f;
    double B_minor_13 = 0.0f;
    double B_minor_21 = 0.0f;
    double B_minor_22 = 0.0f;
    double B_minor_23 = 0.0f;
    double B_minor_31 = 0.0f;
    double B_minor_32 = 0.0f;
    double B_minor_33 = 0.0f;
    // initialize h_lm vector
    double* h_lm = new double[3];
    // initialize h_lm vector to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        h_lm[index_0] = 0.0f;
    }
    // initialize transpose of h_lm vector
    double* h_lm_transpose = new double[3];
    // initialize transpose of h_lm vector to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        h_lm_transpose[index_0] = 0.0f;
    }
    // initialize mi * h_lm_transpose - g
    double* mi_h_lm_trans_minus_g = new double[3];
    // initialize mi * h_lm_transpose - g to zero values
    for (int index_0 = 0; index_0 < 3; index_0++) {
        mi_h_lm_trans_minus_g[index_0] = 0.0f;
    }
    // initialize norm of h_lm vector
    double h_lm_norm = 0.0f;
    // initilize norm of x vector
    double x_norm = 0.0f;
    // initialize F_x value
    double F_x = 0.0f;
    // initialize F_x_new value
    double F_x_new = 0.0f;
    // initialize ro_denominator
    double ro_denominator = 0.0f;
    // intialize ro value - gain ratio
    double ro = 0.0f;

    while (!found_bool && (k < k_max_inp)) {
        // increase iteration by one
        k++;
        // calculate matrix B
        // insert matrix A to matrix B
        for (int index_0 = 0; index_0 < 9; index_0++) {
            B[index_0] = A[index_0];
        }
        B[0] = B[0] + mi;
        B[4] = B[4] + mi;
        B[8] = B[8] + mi;
        // calculate transpose of g
        for (int index_0 = 0; index_0 < 3; index_0++) {
            g_transpose[index_0] = g[index_0];
        }
        // calculate inversion of B
        // calculate minor values
        B_minor_11 = (+1.0f) * (B[4] * B[8] - B[5] * B[7]);
        B_minor_12 = (-1.0f) * (B[3] * B[8] - B[5] * B[6]);
        B_minor_13 = (+1.0f) * (B[3] * B[7] - B[4] * B[6]);
        B_minor_21 = (-1.0f) * (B[1] * B[8] - B[2] * B[7]);
        B_minor_22 = (+1.0f) * (B[0] * B[8] - B[2] * B[6]);
        B_minor_23 = (-1.0f) * (B[0] * B[7] - B[1] * B[6]);
        B_minor_31 = (+1.0f) * (B[1] * B[5] - B[2] * B[4]);
        B_minor_32 = (-1.0f) * (B[0] * B[5] - B[2] * B[3]);
        B_minor_33 = (+1.0f) * (B[0] * B[4] - B[1] * B[3]);
        // calculate adjoint matrix of B
        B_adj[0] = B_minor_11;
        B_adj[1] = B_minor_21;
        B_adj[2] = B_minor_31;
        B_adj[3] = B_minor_12;
        B_adj[4] = B_minor_22;
        B_adj[5] = B_minor_32;
        B_adj[6] = B_minor_13;
        B_adj[7] = B_minor_23;
        B_adj[8] = B_minor_33;
        // calculate determinant value of matrix B
        B_det = B[0] * B_minor_11 + B[1] * B_minor_12 + B[2] * B_minor_13;
        // calculate inversion of matrix B
        for (int index_0 = 0; index_0 < 9; index_0++) {
            B_inv[index_0] = B_adj[index_0] / B_det;
        }
        // calculate h_lm vector
        // initialize h_lm vector to zero values
        for (int index_0 = 0; index_0 < 3; index_0++) {
            h_lm[index_0] = 0.0f;
        }
        for (int index_0 = 0; index_0 < 1; index_0++) {
            for (int index_1 = 0; index_1 < 3; index_1++) {
                for (int index_2 = 0; index_2 < 3; index_2++) {
                    h_lm[index_1 + index_0 * 3] = h_lm[index_1 + index_0 * 3] + (-1.0f) * g_transpose[index_2 + index_0 * 3] * B_inv[index_1 + index_2 * 3];
                }
            }
        }
        // calculate norm of h_lm vector
        // fill norm of h_lm vector with zero value
        h_lm_norm = 0.0f;
        for (int index_0 = 0; index_0 < 3; index_0++) {
            h_lm_norm = h_lm_norm + h_lm[index_0] * h_lm[index_0];
        }
        h_lm_norm = std::sqrt(h_lm_norm);
        // calculate norm of x vector
        // fill norm of x vector with zero value
        x_norm = 0.0f;
        for (int index_0 = 0; index_0 < 3; index_0++) {
            x_norm = x_norm + x[index_0] * x[index_0];
        }
        x_norm = std::sqrt(x_norm);
        // main condition
        if (h_lm_norm <= eps_2_inp * (x_norm + eps_2_inp)) {
            found_bool = true;
        }
        else {
            // calculate vector x_new
            for (int index_0 = 0; index_0 < 3; index_0++) {
                x_new[index_0] = x[index_0] + h_lm[index_0];
            }
            // print iteration result
            //std::cout << k << " " << double(x_new[0]) << " " << double(x_new[1]) << " " << double(x_new[2]) << std::endl;
            // calculate F(x)
            // caclulate function f
            for (int index_0 = 0; index_0 < M_inp; index_0++) {
                f[index_0] = y_data_inp[index_0] - x[0] * std::sin(t_data_inp[index_0] + x[1]) - x[2];
            }
            // calculate F_x value
            // initialize F_x to zero value 
            F_x = 0.0f;
            for (int index_0 = 0; index_0 < M_inp; index_0++) {
                F_x = F_x + f[index_0] * f[index_0];
            }
            F_x = 0.5f * F_x;

            // calculate F(x_new)
            // calculate function f
            for (int index_0 = 0; index_0 < M_inp; index_0++) {
                f[index_0] = y_data_inp[index_0] - x_new[0] * std::sin(t_data_inp[index_0] + x_new[1]) - x_new[2];
            }
            // calculate F_x_new value
            // initialize F_x_new to zero value
            F_x_new = 0.0f;
            for (int index_0 = 0; index_0 < M_inp; index_0++) {
                F_x_new = F_x_new + f[index_0] * f[index_0];
            }
            F_x_new = 0.5f * F_x_new;

            // calculate ro_denominator part
            // initialize transpose of h_lm vector
            for (int index_0 = 0; index_0 < 3; index_0++) {
                h_lm_transpose[index_0] = h_lm[index_0];
            }
            // calculate mi * h_lm_transpose - g
            // initialize mi * h_lm_transpose - g to zero values
            for (int index_0 = 0; index_0 < 3; index_0++) {
                mi_h_lm_trans_minus_g[index_0] = 0.0f;
            }
            for (int index_0 = 0; index_0 < 3; index_0++) {
                mi_h_lm_trans_minus_g[index_0] = mi * h_lm_transpose[index_0] - g[index_0];
            }
            // calculate ro_denominator
            ro_denominator = 0.0f;
            for (int index_0 = 0; index_0 < 1; index_0++) {
                for (int index_1 = 0; index_1 < 1; index_1++) {
                    for (int index_2 = 0; index_2 < 3; index_2++) {
                        ro_denominator = ro_denominator + h_lm[index_2 + index_0 * 3] * mi_h_lm_trans_minus_g[index_1 + index_2 * 1];
                    }
                }
            }
            ro_denominator = 0.5f * ro_denominator;
            // calculate ro value - gain ratio
            ro = (F_x - F_x_new) / ro_denominator;
            if (ro > 0.0f) {
                // insert vector x_new into the vector x
                for (int index_0 = 0; index_0 < 3; index_0++) {
                    x[index_0] = x_new[index_0];
                }
                // fill Jacobian matrix
                for (int index_0 = 0; index_0 < M_inp; index_0++) {
                    J[0 + index_0 * 3] = (-1.0f) * std::sin(t_data_inp[index_0] + x[1]);
                    J[1 + index_0 * 3] = (-1.0f) * x[0] * std::cos(t_data_inp[index_0] + x[1]);
                    J[2 + index_0 * 3] = -1.0f;
                }
                // fill transpose of Jacobian matrix
                for (int index_0 = 0; index_0 < M_inp; index_0++) {
                    for (int index_1 = 0; index_1 < 3; index_1++) {
                        J_transpose[index_0 + index_1 * M_inp] = J[index_1 + index_0 * 3];
                    }
                }
                // calculate A matrix
                // initialize A matrix to zero values
                for (int index_0 = 0; index_0 < 9; index_0++) {
                    A[index_0] = 0.0f;
                }
                // calculate A matrix as J_transpose * J
                // use multiplication of 2D matrices
                for (int index_0 = 0; index_0 < 3; index_0++) {
                    for (int index_1 = 0; index_1 < 3; index_1++) {
                        for (int index_2 = 0; index_2 < M_inp; index_2++) {
                            A[index_1 + index_0 * 3] = A[index_1 + index_0 * 3] + J_transpose[index_2 + index_0 * M_inp] * J[index_1 + index_2 * 3];
                        }
                    }
                }
                // calculate f function
                // fill f function
                for (int index_0 = 0; index_0 < M_inp; index_0++) {
                    f[index_0] = y_data_inp[index_0] - x[0] * std::sin(t_data_inp[index_0] + x[1]) - x[2];
                }
                // calculate transpose of f
                // fill transpose of f
                for (int index_0 = 0; index_0 < M_inp; index_0++) {
                    f_transpose[index_0] = f[index_0];
                }
                for (int index_0 = 0; index_0 < 3; index_0++) {
                    g[index_0] = 0.0f;
                }
                // calculate g as J_transpose * f_transpose
                for (int index_0 = 0; index_0 < 3; index_0++) {
                    for (int index_1 = 0; index_1 < 1; index_1++) {
                        for (int index_2 = 0; index_2 < M_inp; index_2++) {
                            g[index_1 + index_0 * 1] = g[index_1 + index_0 * 1] + J_transpose[index_2 + index_0 * M_inp] * f_transpose[index_1 + index_2 * 1];
                        }
                    }
                }
                // calculate norm of g
                // initialize norm of g
                g_norm = 0.0f;
                for (int index_0 = 0; index_0 < 3; index_0++) {
                    g_norm = g_norm + g[index_0] * g[index_0];
                }
                g_norm = std::sqrt(g_norm);
                // calculate boolean variable
                found_bool = (g_norm <= eps_1_inp);
                // calculate mi
                double value_1 = double(0.33333333333333333333f);
                double value_2 = double(1.0f - std::pow((2.0f * ro - 1), 3.0f));
                double max_value = 0.0f;
                if (value_1 >= value_2) {
                    max_value = value_1;
                }
                else {
                    max_value = value_2;
                }
                mi = mi * max_value;
                // define ni
                ni = 2;
            }
            else {
                // calculate mi
                mi = mi * double(ni);
                // calculate ni
                ni = 2 * ni;
            }
        }
    }

    ////std::cout << x_new << std::endl;
    //// convert phase shift from fitting to interval (0, 2*pi)
    //if (x_new[1] > 0.0f && x_new[0] > 0.0f) {
    //    x_new[1] = x_new[1] - (2.0f * M_PI) * int(x_new[1] / (2 * M_PI));
    //}
    //else if (x_new[1] < 0.0f && x_new[0] > 0.0f) {
    //    x_new[1] = x_new[1] - (2.0f * M_PI) * (int(x_new[1] / (2 * M_PI)) - 1);
    //}
    //else if (x_new[1] > 0.0f && x_new[0] < 0.0f) {
    //    x_new[0] = (-1.0f) * x_new[0];
    //    x_new[1] = x_new[1] + 1.0f * M_PI;
    //    x_new[1] = x_new[1] - (2.0f * M_PI) * int(x_new[1] / (2 * M_PI));
    //}
    //else if (x_new[1] < 0.0f && x_new[0] < 0.0f) {
    //    x_new[0] = (-1.0f) * x_new[0];
    //    x_new[1] = x_new[1] - 1.0f * M_PI;
    //    x_new[1] = x_new[1] - (2.0f * M_PI) * (int(x_new[1] / (2 * M_PI)) - 1);
    //}
    //else {
    //    x_new[1] = 0.0f;
    //}
    ////std::cout << x_new << std::endl;

    // store fitting results to output 1D double array
    x_fit_outp[0] = x_new[0];
    x_fit_outp[1] = x_new[1];
    x_fit_outp[2] = x_new[2];

    delete[] J;
    delete[] J_transpose;
    delete[] A;
    delete[] f;
    delete[] f_transpose;
    delete[] g;
    delete[] g_transpose;
    delete[] x;
    delete[] x_new;
    delete[] B;
    delete[] B_inv;
    delete[] B_adj;
    delete[] h_lm;
    delete[] h_lm_transpose;
    delete[] mi_h_lm_trans_minus_g;

    return 0;
}

__global__ void XTI_kernel(unsigned short int* image_buffer_1D_fg_GPU_knl, unsigned short int* image_buffer_1D_bg_GPU_knl, unsigned int* no_pixels_GPU_knl, unsigned int* M_fg_GPU_knl, unsigned int* M_bg_GPU_knl, double* phase_step_fg_GPU_knl, double* phase_step_bg_GPU_knl, double* dph_image_GPU_knl, double* abs_image_GPU_knl, double* vis_image_GPU_knl)
{
    int index_pixel = threadIdx.x + blockIdx.x * blockDim.x;

    int no_pixels_GPU_kernel = int(*no_pixels_GPU_knl);
    const unsigned int M_fg_GPU_kernel = *M_fg_GPU_knl;
    const unsigned int M_bg_GPU_kernel = *M_bg_GPU_knl;
    double phase_step_fg_GPU_kernel = *phase_step_fg_GPU_knl;
    double phase_step_bg_GPU_kernel = *phase_step_bg_GPU_knl;

    double phase_buffer_fg = 0.0f;
    double amp_buffer_fg = 0.0f;
    double offset_buffer_fg = 0.0f;

    double phase_buffer_bg = 0.0f;
    double amp_buffer_bg = 0.0f;
    double offset_buffer_bg = 0.0f;

    // define intial parameters of sinusoidal function for fitting
    // intial amplitude
    double x01 = 1.0f;
    // initial phase shift
    double x02 = double(M_PI / 2.0f);
    // initial offset
    double x03 = 0.0f;
    // define initial parameter vector
    double* x0 = new double[3];
    // fill initial parameters vector
    x0[0] = x01;
    x0[1] = x02;
    x0[2] = x03;
    // initialize variables for calculation initial parameters x0
    // for foreground
    double y_data_fg_max = 0.0f;
    double y_data_fg_min = 0.0f;
    // for background
    double y_data_bg_max = 0.0f;
    double y_data_bg_min = 0.0f;
    // define input t_data 1D array for foreground, on the t axis 
    double* t_data_fg = new double[M_fg_GPU_kernel];
    for (unsigned int index_0 = 0; index_0 < M_fg_GPU_kernel; index_0++) {
        //t_data_fg[index_0] = double(index_0) * (double(2.0f * M_PI) / double(M_fg_GPU_kernel));
        t_data_fg[index_0] = double(index_0) * phase_step_fg_GPU_kernel;
    }
    // define input t_data 1D array for background, on the t axis
    double* t_data_bg = new double[M_bg_GPU_kernel];
    for (unsigned int index_0 = 0; index_0 < M_bg_GPU_kernel; index_0++) {
        //t_data_bg[index_0] = double(index_0) * (double(2.0f * M_PI) / double(M_bg_GPU_kernel));
        t_data_bg[index_0] = double(index_0) * phase_step_bg_GPU_kernel;
    }
    // define input y_data 1D array for foreground
    double* y_data_fg = new double[M_fg_GPU_kernel];
    // define input y_data 1D array for background
    double* y_data_bg = new double[M_bg_GPU_kernel];
    // maximum number of iterations in fitting
    int k_max = 1000;
    // auxiliar fitting variable epsilon 1
    double eps_1 = 1.0E-8f;
    // auxiliar fitting variable epsilon 2
    double eps_2 = 1.0E-8f;
    // auxiliar fitting variable tau
    double tau = 1.0E-3f;
    // create output 1D array where fitting results will be stored
    double* x_fit = new double[3];

    // calculate foreground values in single pixel
    x0[2] = 0.0f;
    for (int index_0 = 0; index_0 < M_fg_GPU_kernel; index_0++) {
        y_data_fg[index_0] = double(image_buffer_1D_fg_GPU_knl[index_pixel + index_0 * no_pixels_GPU_kernel]);
        if (index_0 == 0) {
            y_data_fg_max = y_data_fg[0];
            y_data_fg_min = y_data_fg[0];
        }
        else {
            if (y_data_fg_max >= y_data_fg[index_0]) {
                // do nothing
            }
            else {
                y_data_fg_max = y_data_fg[index_0];
            }
            if (y_data_fg_min <= y_data_fg[index_0]) {
                // do nothing
            }
            else {
                y_data_fg_min = y_data_fg[index_0];
            }
        }
        x0[2] += y_data_fg[index_0] / double(M_fg_GPU_kernel);
    }
    x0[0] = (y_data_fg_max - y_data_fg_min) / 2;
    levmar_sinus(t_data_fg, y_data_fg, M_fg_GPU_kernel, x0, x_fit, k_max, eps_1, eps_2, tau);
    phase_buffer_fg = x_fit[1];
    amp_buffer_fg = x_fit[0];
    offset_buffer_fg = x_fit[2];

    // calculate background values in single pixel
    x0[2] = 0.0f;
    for (int index_0 = 0; index_0 < M_bg_GPU_kernel; index_0++) {
        y_data_bg[index_0] = double(image_buffer_1D_bg_GPU_knl[index_pixel + index_0 * no_pixels_GPU_kernel]);
        if (index_0 == 0) {
            y_data_bg_max = y_data_bg[0];
            y_data_bg_min = y_data_bg[0];
        }
        else {
            if (y_data_bg_max >= y_data_bg[index_0]) {
                // do nothing
            }
            else {
                y_data_bg_max = y_data_bg[index_0];
            }
            if (y_data_bg_min <= y_data_bg[index_0]) {
                // do nothing
            }
            else {
                y_data_bg_min = y_data_bg[index_0];
            }
        }
        x0[2] += y_data_bg[index_0] / double(M_bg_GPU_kernel);
    }
    x0[0] = (y_data_bg_max - y_data_bg_min) / 2;
    levmar_sinus(t_data_bg, y_data_bg, M_bg_GPU_kernel, x0, x_fit, k_max, eps_1, eps_2, tau);
    phase_buffer_bg = x_fit[1];
    amp_buffer_bg = x_fit[0];
    offset_buffer_bg = x_fit[2];

    // calculate differential phase image or dph image
    dph_image_GPU_knl[index_pixel] = phase_buffer_fg - phase_buffer_bg;
    // calculate absorption image or dph image
    abs_image_GPU_knl[index_pixel] = offset_buffer_fg / offset_buffer_bg;
    // calculate visibility image or dph image
    vis_image_GPU_knl[index_pixel] = (amp_buffer_fg / offset_buffer_fg) / (amp_buffer_bg / offset_buffer_bg);

    // delete buffers created for fitting
    delete[] x0;
    delete[] t_data_fg;
    delete[] t_data_bg;
    delete[] y_data_fg;
    delete[] y_data_bg;
    delete[] x_fit;
}

int main()
{
    // define path to folder with all foreground data or all subfolders
    string path_to_fg_folder("d:/XTI_Momose_lab/BL28B2_2017A/sort_data/pp/fg/");
    // define path to folder with all background data or all subfolders
    string path_to_bg_folder("d:/XTI_Momose_lab/BL28B2_2017A/sort_data/bg/");
    // print path to folder with all foreground folders
    std::cout << path_to_fg_folder << "\n";
    // print path to folder with all background folders
    std::cout << path_to_bg_folder << "\n";

    // define path to output folder with output differential phase (dph) images
    string path_to_output_folder("d:/XTI_Momose_lab/BL28B2_2017A/sort_data/pp_dph_abs_vis_Momose_sinus_fitting_low_level_1D_GPU/");

    // output image name root - differential phase image or dph image
    string image_output_dph_name_root = "dph";
    // output image name root - absorption image or abs image
    string image_output_abs_name_root = "abs";
    // output image name root - visibility image or vis image
    string image_output_vis_name_root = "vis";
    // final ouput image name - differential phase image or dph image
    string image_output_dph_name;
    // final ouput image name - absorption image or abs image
    string image_output_abs_name;
    // final ouput image name - visibility image or vis image
    string image_output_vis_name;
    // extension of output image
    string image_output_extension = ".raw";

    // define size of the raw unsigned 16 bit images
    const unsigned int no_cols = 1536; // in pixels, in horizontal direction
    const unsigned int no_rows = 512; // in pixels, in vertical direction
    // total number of pixels in single image
    const unsigned int no_pixels = no_cols * no_rows;
    // total number of bytes in single image, we consider 16 bit values per pixel = 2 bytes
    //const unsigned int no_bytes = 2 * no_pixels;

    // define number of initial and final subfolder for foreground
    unsigned int no_subfolder_fg_initial = 1;
    unsigned int no_subfolder_fg_final = 1;

    // define number of initial and final folder for background
    //unsigned int no_subfolder_bg_initial = 1;
    //unsigned int no_subfolder_bg_final = 1;

    // number of digits in subfolder name for foreground
    string::size_type no_subfolder_digits_fg = 6;
    // number of digits in subfolder name for background
    string::size_type no_subfolder_digits_bg = 6;

    // number of steps in fringe scanning technique
    const unsigned int M = 5;

    // calculate differential phase image for foreground
    // fringe scanning defined from initial value
    const unsigned int M_fg_initial = 1;
    // fringe scanning defined to final value
    const unsigned int M_fg_final = M;
    // number of steps in fringe scanning for foreground
    const unsigned int M_fg = M_fg_final - M_fg_initial + 1;
    const unsigned int N_fg = M_fg * no_pixels;

    // calculate differential phase image for background
    // fringe scanning defined from initial value
    const unsigned int M_bg_initial = 1;
    // fringe scanning defined to final value
    const unsigned int M_bg_final = M;
    // number of steps in fringe scanning for background
    const unsigned int M_bg = M_bg_final - M_bg_initial + 1;
    const unsigned int N_bg = M_bg * no_pixels;

    // define root name of images for foreground
    string root_image_name_fg("a");
    // define root name of images for background
    string root_image_name_bg("a");

    // number of digits in image name for foreground
    string::size_type no_image_digits_fg = 6;
    // number of digits in image name for background
    string::size_type no_image_digits_bg = 6;

    // define image extensions
    // image extension for foreground
    string image_extension_fg = ".raw";
    // image extension for background
    string image_extension_bg = ".raw";

    // allocate image buffer for foreground
    auto image_buffer_fg = new unsigned short int[no_pixels][M_fg];
    // allocate image buffer for background
    auto image_buffer_bg = new unsigned short int[no_pixels][M_fg];
    // allocate image buffer as 1D array for foreground
    // to copy M_fg foreground images to the GPU
    unsigned short int* image_buffer_1D_fg = new unsigned short int[M_fg * no_pixels];
    // allocate image buffer as 1D array for background
    // to copy M_bg background images to the GPU
    unsigned short int* image_buffer_1D_bg = new unsigned short int[M_bg * no_pixels];

    // allocate memory for differential phase image
    double* dph_image = new double[no_pixels];
    // allocate memory for absorption image
    double* abs_image = new double[no_pixels];
    // allocate memory for visibility image
    double* vis_image = new double[no_pixels];

    // define phase for foreground
    double phase_step_fg = (2 * M_PI) / double(M_fg);
    // define phase_step for background
    double phase_step_bg = (2 * M_PI) / double(M_bg);

    // auxiliary variables for iteration through subfolder name for foreground
    string subfolder_name(no_subfolder_digits_fg, '0');
    string subfolder_number = "";
    string::size_type counter_digits = 0;
    string::size_type difference = 0;
    string::size_type counter = 0;
    string path_to_fg_subfolder = "";

    // auxiliary variables for iteration through M_fg images
    int counter_image = 0;
    string image_name = root_image_name_fg;
    string image_name_number(no_image_digits_fg, '0');
    string image_number = "";
    // counter_digits, difference and counter variables are taken from iterations through subfolders
    string path_to_fg_image = "";

    // auxiliary variables for iteration through subfolder name for background
    string path_to_bg_subfolder = "";

    // auxiliary variables for iteration through M_bg images
    image_name = root_image_name_bg;
    image_name_number = string(no_image_digits_bg, '0');
    image_number = "";
    // counter_digits, difference and counter variables are taken from iterations through subfolders
    string path_to_bg_image = "";

    // declare auxiliary variable for output image
    string path_to_output_image = "";

    // declare auxiliary variable for output dph image
    string path_to_output_dph_image = "";
    // declare auxiliary variable for output abs image
    string path_to_output_abs_image = "";
    // declare auxiliary variable for output vis image
    string path_to_output_vis_image = "";

    //*****************************************************
    // variables for the GPU CUDA
    //*****************************************************
    // initialize varaible for number of columns of the images on the GPU
    unsigned int* no_cols_GPU = nullptr;
    // initialize varaible for number of rows of the images on the GPU
    unsigned int* no_rows_GPU = nullptr;
    // initialize variable for number of pixels on the GPU
    unsigned int* no_pixels_GPU = nullptr;
    // initialize variable for number of steps in fringe scanning for foreground on the GPU
    unsigned int* M_fg_GPU = nullptr;
    // initialize variable for number of steps in fringe scanning for background on the GPU
    unsigned int* M_bg_GPU = nullptr;
    // initialize 1D array for M_fg foreground images on the GPU
    unsigned short int* image_buffer_1D_fg_GPU = nullptr;
    // initialize 1D array for M_bg background images on the GPU
    unsigned short int* image_buffer_1D_bg_GPU = nullptr;
    // initialize phase step in fringe scanning for foreground on the GPU
    double* phase_step_fg_GPU = nullptr;
    // initialize phase step in fringe scanning for foreground on the GPU
    double* phase_step_bg_GPU = nullptr;
    // initialize output 1D arrays
    // initialize output 1D array for differential phase image (dph)
    double* dph_image_GPU = nullptr;
    // initialize output 1D array for absorption image (abs)
    double* abs_image_GPU = nullptr;
    // initialize output 1D array for visibility image (vis)
    double* vis_image_GPU = nullptr;
    //*****************************************************
    // end
    //*****************************************************

    //*****************************************************
    // fill pointers with constant values
    //*****************************************************
    unsigned int* no_cols_ptr = new unsigned int;
    *no_cols_ptr = no_cols;
    unsigned int* no_rows_ptr = new unsigned int;
    *no_rows_ptr = no_rows;
    unsigned int* no_pixels_ptr = new unsigned int;
    *no_pixels_ptr = no_pixels;
    unsigned int* M_fg_ptr = new unsigned int;
    *M_fg_ptr = M_fg;
    unsigned int* M_bg_ptr = new unsigned int;
    *M_bg_ptr = M_bg;
    double* phase_step_fg_ptr = new double;
    *phase_step_fg_ptr = phase_step_fg;
    double* phase_step_bg_ptr = new double;
    *phase_step_bg_ptr = phase_step_bg;
    //*****************************************************
    // end
    //*****************************************************

    //*****************************************************
    // define number of therads and blocks
    //*****************************************************
    int no_threads = 512;
    int no_blocks = 1536;
    //*****************************************************
    // end
    //*****************************************************

    // go through all foreground subfolders and foreground images
    for (unsigned int index_0 = no_subfolder_fg_initial; index_0 <= no_subfolder_fg_final; index_0++) {
        // start to measure elapsed time at the beginning
        std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
        // initialize subfolder name to "000000"
        subfolder_name = string(no_subfolder_digits_fg, '0');
        // typcast integer value to string, convert integer value to string
        subfolder_number = std::to_string(index_0);
        // initialize digits counter
        counter_digits = subfolder_number.size();
        // initialize difference
        difference = no_subfolder_digits_fg - counter_digits;
        // initialize counter
        counter = 0;
        // generate subfolder name
        for (string::size_type index_1 = difference; index_1 < no_subfolder_digits_fg; index_1++) {
            subfolder_name[index_1] = subfolder_number[counter];
            counter++;
        }
        // generate path to foreground subfolder 
        path_to_fg_subfolder = path_to_fg_folder + subfolder_name;
        // print final path to foreground subfolder
        std::cout << path_to_fg_subfolder << "\n";
        // initilize counter for one of M images
        counter_image = 0;
        // open images in foreground subfolder
        for (unsigned int index_2 = M_fg_initial; index_2 <= M_fg_final; index_2++) {
            // initialize image name to root value
            image_name = root_image_name_fg;
            // initialize image number to "000000"
            image_name_number = string(no_image_digits_fg, '0');
            // typcast integer value to string, convert integer value to string
            image_number = std::to_string(index_2);
            // initialize digits counter
            counter_digits = image_number.size();
            // initialize difference
            difference = no_image_digits_fg - counter_digits;
            // initialize counter
            counter = 0;
            // generate image name number
            for (string::size_type index_3 = difference; index_3 < no_image_digits_fg; index_3++) {
                image_name_number[index_3] = image_number[counter];
                counter++;
            }
            // concatenate root image value and image number
            image_name += image_name_number;
            // generate path to foreground image 
            path_to_fg_image = path_to_fg_subfolder + "/" + image_name + image_extension_fg;
            // print path to image for foregrouund
            //std::cout << path_to_fg_image << "\n";
            //************************************************************************************
            // read binary images
            //************************************************************************************
            ifstream raw_image(path_to_fg_image, ios::out | ios::binary);
            /*streampos begin, end;
            begin = raw_image.tellg();
            raw_image.seekg(0, ios::end);
            end = raw_image.tellg();*/
            //std::cout << "Size of the raw image is: " << (end - begin) << " bytes.\n";
            if (raw_image.is_open())
            {
                //unsigned int counter_pixel = 0;
                //raw_image.seekg(0, ios::beg);
                //while (raw_image.read(reinterpret_cast<char*>(&image_buffer_fg[counter_pixel][counter_image]), sizeof(uint16_t))) { // Read 16-bit integer values from file
                //    counter_pixel++;
                //}
                //raw_image.close();
                for (unsigned int counter_pixel = 0; counter_pixel < no_pixels; counter_pixel++) {
                    raw_image.read((char*)&image_buffer_fg[counter_pixel][counter_image], sizeof(unsigned short int));
                }
                raw_image.close();
            }
            else {
                std::cout << "Warning: Unable to open raw image file!!!" << "\n";
            }
            //************************************************************************************
            // end of reading of binary images
            //************************************************************************************
            // increase image counter by one
            counter_image++;
        }

        // go through background subfolder and background images
        // initialize subfolder name to "000000"
        subfolder_name = string(no_subfolder_digits_bg, '0');
        // typcast integer value to string, convert integer value to string
        subfolder_number = std::to_string(index_0);
        // initialize digits counter
        counter_digits = subfolder_number.size();
        // initialize difference
        difference = no_subfolder_digits_bg - counter_digits;
        // initialize counter
        counter = 0;
        // generate subfolder name
        for (string::size_type index_1 = difference; index_1 < no_subfolder_digits_bg; index_1++) {
            subfolder_name[index_1] = subfolder_number[counter];
            counter++;
        }
        // generate path to background subfolder 
        path_to_bg_subfolder = path_to_bg_folder + subfolder_name;
        // print final path to background subfolder
        std::cout << path_to_bg_subfolder << "\n";
        // initilize counter for one of M images
        counter_image = 0;
        // open images in background subfolder
        for (unsigned int index_2 = M_bg_initial; index_2 <= M_bg_final; index_2++) {
            // initialize image name to root value
            image_name = root_image_name_bg;
            // initialize image number to "000000"
            image_name_number = string(no_image_digits_bg, '0');
            // typcast integer value to string, convert integer value to string
            image_number = std::to_string(index_2);
            // initialize digits counter
            counter_digits = image_number.size();
            // initialize difference
            difference = no_image_digits_bg - counter_digits;
            // initialize counter
            counter = 0;
            // generate image name number
            for (string::size_type index_3 = difference; index_3 < no_image_digits_bg; index_3++) {
                image_name_number[index_3] = image_number[counter];
                counter++;
            }
            // concatenate root image value and image number
            image_name += image_name_number;
            // generate path to background image 
            path_to_bg_image = path_to_bg_subfolder + "/" + image_name + image_extension_bg;
            // print path to image for background
            //std::cout << path_to_bg_image << "\n";
            //************************************************************************************
            // read binary images
            //************************************************************************************
            ifstream raw_image(path_to_bg_image, ios::out | ios::binary);
            /*streampos begin, end;
            begin = raw_image.tellg();
            raw_image.seekg(0, ios::end);
            end = raw_image.tellg();*/
            //std::cout << "Size of the raw image is: " << (end - begin) << " bytes.\n";
            if (raw_image.is_open())
            {
                //unsigned int counter_pixel = 0;
                //raw_image.seekg(0, ios::beg);
                //while (raw_image.read(reinterpret_cast<char*>(&image_buffer_bg[counter_pixel][counter_image]), sizeof(uint16_t))) { // Read 16-bit integer values from file
                //    counter_pixel++;
                //}
                //raw_image.close();
                raw_image.seekg(0, ios::beg);
                for (unsigned int counter_pixel = 0; counter_pixel < no_pixels; counter_pixel++) {
                    raw_image.read((char*)&image_buffer_bg[counter_pixel][counter_image], sizeof(unsigned short int));
                }
                raw_image.close();
            }
            else {
                std::cout << "Warning: Unable to open raw image file!!!" << "\n";
            }
            //************************************************************************************
            // end of reading of binary images
            //************************************************************************************
            // increase image counter by one
            counter_image++;
        }

        // transfer images for foreground to 1D array
        // this 1D array will be later transferred to the GPU, it is data preparation for GPU
        for (unsigned int index_4 = 0; index_4 < M_fg; index_4++) {
            for (unsigned int index_5 = 0; index_5 < no_pixels; index_5++) {
                image_buffer_1D_fg[index_5 + index_4 * no_pixels] = image_buffer_fg[index_5][index_4];
            }
        }
        // transfer images for background to 1D array
        // this 1D array will be later transferred to the GPU, it is data preparation for GPU
        // this 1D array will be later transferred to the GPU, it is data preparation for GPU
        for (unsigned int index_4 = 0; index_4 < M_bg; index_4++) {
            for (unsigned int index_5 = 0; index_5 < no_pixels; index_5++) {
                image_buffer_1D_bg[index_5 + index_4 * no_pixels] = image_buffer_bg[index_5][index_4];
            }
        }

        // allocate memory on the GPU
        // allocate memory on the GPU for number of columns in the images
        hipMalloc((void**)&no_cols_GPU, sizeof(unsigned int));
        // allocate memory on the GPU for number of rows in the images
        hipMalloc((void**)&no_rows_GPU, sizeof(unsigned int));
        // allocate memory on the GPU for number of pixels in the images
        hipMalloc((void**)&no_pixels_GPU, sizeof(unsigned int));
        // allocate memory for number of steps in fringe scanning for foreground
        hipMalloc((void**)&M_fg_GPU, sizeof(unsigned int));
        // allocate memory for number of steps in fringe scanning for background
        hipMalloc((void**)&M_bg_GPU, sizeof(unsigned int));
        // allocate memory for 1D array to store unsigned 16 bit integer raw images for foreground
        hipMalloc((void**)&image_buffer_1D_fg_GPU, N_fg * sizeof(unsigned short int));
        // allocate memory for 1D array to store unsigned 16 bit integer raw images for background
        hipMalloc((void**)&image_buffer_1D_bg_GPU, N_bg * sizeof(unsigned short int));
        // allocate memory for storing phase step for foreground
        hipMalloc((void**)&phase_step_fg_GPU, sizeof(double));
        // allocate memory for storing phase step for foreground
        hipMalloc((void**)&phase_step_bg_GPU, sizeof(double));
        // allocate memory for differential phase (dph) image calculated on the GPU
        hipMalloc((void**)&dph_image_GPU, no_pixels * sizeof(double));
        // allocate memory for absorption (abs) image calculated on the GPU
        hipMalloc((void**)&abs_image_GPU, no_pixels * sizeof(double));
        // allocate memory for visibility (vis) image calculated on the GPU
        hipMalloc((void**)&vis_image_GPU, no_pixels * sizeof(double));

        // copy all data to the GPU
        hipMemcpy(no_cols_GPU, no_cols_ptr, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(no_rows_GPU, no_rows_ptr, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(no_pixels_GPU, no_pixels_ptr, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(M_fg_GPU, M_fg_ptr, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(M_bg_GPU, M_bg_ptr, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(image_buffer_1D_fg_GPU, image_buffer_1D_fg, N_fg * sizeof(unsigned short int), hipMemcpyHostToDevice);
        hipMemcpy(image_buffer_1D_bg_GPU, image_buffer_1D_bg, N_bg * sizeof(unsigned short int), hipMemcpyHostToDevice);
        hipMemcpy(phase_step_fg_GPU, phase_step_fg_ptr, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(phase_step_bg_GPU, phase_step_bg_ptr, sizeof(double), hipMemcpyHostToDevice);

        // perform kernel calculation or calculation on the GPU
        XTI_kernel << < no_blocks, no_threads >> > (image_buffer_1D_fg_GPU, image_buffer_1D_bg_GPU, no_pixels_GPU, M_fg_GPU, M_bg_GPU, phase_step_fg_GPU, phase_step_bg_GPU, dph_image_GPU, abs_image_GPU, vis_image_GPU);

        // copy results from the GPU to the CPU
        hipMemcpy(dph_image, dph_image_GPU, no_pixels * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(abs_image, abs_image_GPU, no_pixels * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(vis_image, vis_image_GPU, no_pixels * sizeof(double), hipMemcpyDeviceToHost);

        // free the memory allocated on the GPU
        hipFree(no_cols_GPU);
        hipFree(no_rows_GPU);
        hipFree(no_pixels_GPU);
        hipFree(M_fg_GPU);
        hipFree(M_bg_GPU);
        hipFree(image_buffer_1D_fg_GPU);
        hipFree(image_buffer_1D_bg_GPU);
        hipFree(phase_step_fg_GPU);
        hipFree(phase_step_bg_GPU);
        hipFree(dph_image_GPU);
        hipFree(abs_image_GPU);
        hipFree(vis_image_GPU);

        // define name for output dph image for current subfolder
        image_output_dph_name = image_output_dph_name_root + "_" + subfolder_name + image_output_extension;
        // define name for output abs image for current subfolder
        image_output_abs_name = image_output_abs_name_root + "_" + subfolder_name + image_output_extension;
        // define name for output vis image for current subfolder
        image_output_vis_name = image_output_vis_name_root + "_" + subfolder_name + image_output_extension;
        // define path to the output dph image
        path_to_output_dph_image = path_to_output_folder + image_output_dph_name;
        // define path to the output abs image
        path_to_output_abs_image = path_to_output_folder + image_output_abs_name;
        // define path to the output vis image
        path_to_output_vis_image = path_to_output_folder + image_output_vis_name;
        // write differential phase (dph) image
        // set for output, binary data, trunc
        fstream output_dph_image(path_to_output_dph_image, ios::out | ios::binary | ios::trunc);
        if (output_dph_image.is_open())
        {
            // set pointer to the beginning of the image
            output_dph_image.seekg(0, ios::beg);
            for (unsigned int index_11 = 0; index_11 < no_pixels; index_11++) {
                output_dph_image.write((char*)&dph_image[index_11], sizeof(double));
            }
            output_dph_image.close();
        }
        else {
            std::cout << "Warning: Unable to open dph image file!!!" << "\n";
        }
        // write absorption (abs) image
        // set for output, binary data, trunc
        fstream output_abs_image(path_to_output_abs_image, ios::out | ios::binary | ios::trunc);
        if (output_abs_image.is_open())
        {
            // set pointer to the beginning of the image
            output_abs_image.seekg(0, ios::beg);
            for (unsigned int index_11 = 0; index_11 < no_pixels; index_11++) {
                output_abs_image.write((char*)&abs_image[index_11], sizeof(double));
            }
            output_abs_image.close();
        }
        else {
            std::cout << "Warning: Unable to open abs image file!!!" << "\n";
        }
        // write visibility (vis) image
        // set for output, binary data, trunc
        fstream output_vis_image(path_to_output_vis_image, ios::out | ios::binary | ios::trunc);
        if (output_vis_image.is_open())
        {
            // set pointer to the beginning of the image
            output_vis_image.seekg(0, ios::beg);
            for (unsigned int index_11 = 0; index_11 < no_pixels; index_11++) {
                output_vis_image.write((char*)&vis_image[index_11], sizeof(double));
            }
            output_vis_image.close();
        }
        else {
            std::cout << "Warning: Unable to open vis image file!!!" << "\n";
        }

        // stop to measure elapsed time at the end
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        // print elapsed time in milliseconds, microseconds and nanoseconds
        //std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::seconds>(end - begin).count() << "[seconds]" << std::endl;
        std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[millisec]" << std::endl;
        //std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[microsec]" << std::endl;
        //std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[nanosec]" << std::endl;
    }

    // delete image buffer for foreground
    delete[] image_buffer_fg;
    // delete image buffer for background
    delete[] image_buffer_bg;
    // delete image buffer for foreground
    delete[] image_buffer_1D_fg;
    // delete image buffer for background
    delete[] image_buffer_1D_bg;
    // delete buffer for differential phase image
    delete[] dph_image;
    // delete buffer for absorption image
    delete[] abs_image;
    // delete buffer for visibility image
    delete[] vis_image;

    // delete host pointers used to transfer data to the GPU
    delete no_cols_ptr;
    delete no_rows_ptr;
    delete no_pixels_ptr;
    delete M_fg_ptr;
    delete M_bg_ptr;
    delete phase_step_fg_ptr;
    delete phase_step_bg_ptr;

    // delete GPU or device pointers
    /*delete no_cols_GPU;
    delete no_rows_GPU;
    delete no_pixels_GPU;
    delete M_fg_GPU;
    delete M_bg_GPU;
    delete image_buffer_1D_fg_GPU;
    delete image_buffer_1D_bg_GPU;
    delete phase_step_fg_GPU;
    delete phase_step_bg_GPU;
    delete dph_image_GPU;
    delete abs_image_GPU;
    delete vis_image_GPU;*/

    return 0;
}
